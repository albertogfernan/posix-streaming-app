#include "hip/hip_runtime.h"
#include "kernel_movementfilter_gpu.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE_1D 256

unsigned char *frame_data_device, *ref_data_device;
int * reduction;
int * devReduction;

void initCUDA_MF()
{
	int size;

	hipMalloc((void**) &ref_data_device, MAX_BUFFER);
	hipMemset(ref_data_device, 0, MAX_BUFFER);
	hipMalloc((void**) &frame_data_device, MAX_BUFFER);
	hipMemset(frame_data_device, 0, MAX_BUFFER);

	size = (frame_size / 4) / BLOCK_SIZE_1D;
	reduction = (int*) malloc(size);
	hipMalloc(&devReduction, size);
	hipMemset(devReduction, 0, size);
	hipHostRegister(reduction, size, hipHostRegisterDefault);

	initCUDAMeasures();
}


void freeCUDA_MF()
{
	hipFree(ref_data_device);
	hipFree(frame_data_device);

	hipFree(devReduction);
	hipHostUnregister(reduction);
	free(reduction);

	freeCUDAMeasures();

	hipDeviceReset();
}


template<unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
	if (blockSize >= 64)
		sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32)
		sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16)
		sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8)
		sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4)
		sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2)
		sdata[tid] += sdata[tid + 1];
}

template<unsigned int blockSize>
__global__ void reduce6_RGBx(unsigned char *g_idata, int *g_odata, int n) {
	extern __shared__ int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) {
		sdata[tid] += ((uchar4*)g_idata)[i].x + ((uchar4*)g_idata)[i + blockSize].x;
		i += gridSize;
	}

	__syncthreads();

	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32)
		warpReduce<blockSize>(sdata, tid);

	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}


template<unsigned int blockSize>
__global__ void reduce6_RGB(unsigned char *g_idata, int *g_odata, int n) {
	extern __shared__ int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) {
		sdata[tid] += ((uchar3*)g_idata)[i].x + ((uchar3*)g_idata)[i + blockSize].x;
		i += gridSize;
	}

	__syncthreads();

	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32)
		warpReduce<blockSize>(sdata, tid);

	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}


template<unsigned int blockSize>
__global__ void reduce6_int(int *g_idata, int *g_odata, int n) {
	extern __shared__ int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) {
		sdata[tid] += g_idata[i] + g_idata[i + blockSize];
		i += gridSize;
	}

	__syncthreads();

	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32)
		warpReduce<blockSize>(sdata, tid);

	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}


__global__ void convert2greyscale_cuda_kernel_RGBx(int width, int height, int pixel_stride, int row_stride, unsigned char *dataIn, unsigned char *dataOut, unsigned char *dataRef)
{
	int absolute_position;
	int luma;

	absolute_position = (blockIdx.x * blockDim.x) + threadIdx.x;

	uchar4 a=((uchar4*)dataIn)[absolute_position];
	uchar4 aref=((uchar4*)dataRef)[absolute_position];

	luma = (a.x + a.y + a.z) / 3;

	a.x = abs(luma - aref.x);
	a.y = abs(luma - aref.y);
	a.z = abs(luma - aref.z);

	aref.x = luma;
	aref.y = luma;
	aref.z = luma;

	((uchar4*)dataOut)[absolute_position] = a;
	((uchar4*)dataRef)[absolute_position] = aref;
}


__global__ void convert2greyscale_cuda_kernel_RGB(int width, int height, int pixel_stride, int row_stride, unsigned char *dataIn, unsigned char *dataOut, unsigned char *dataRef)
{
	int absolute_position;
	int luma;

	absolute_position = (blockIdx.x * blockDim.x) + threadIdx.x;

	uchar3 a=((uchar3*)dataIn)[absolute_position];
	uchar3 aref=((uchar3*)dataRef)[absolute_position];

	luma = (a.x + a.y + a.z) / 3;

	a.x = abs(luma - aref.x);
	a.y = abs(luma - aref.y);
	a.z = abs(luma - aref.z);

	aref.x = luma;
	aref.y = luma;
	aref.z = luma;

	((uchar3*)dataOut)[absolute_position] = a;
	((uchar3*)dataRef)[absolute_position] = aref;
}



long long convert2greyscale_cuda_kernel(unsigned char *data, unsigned char *dataRef, int height_parm)
{
	long long pixel_diff_gpgpu = 0;
	unsigned int row_stride = width * bpp;
	int true_size = row_stride * height_parm;
	hipHostRegister(data, true_size, hipHostRegisterDefault);
	hipMemcpy(frame_data_device, data, true_size, hipMemcpyHostToDevice);

	const dim3 blockSize(BLOCK_SIZE_1D, 1, 1);
	const dim3 gridSize((true_size / bpp) / BLOCK_SIZE_1D, 1, 1);
	const dim3 gridSize2(gridSize.x / BLOCK_SIZE_1D, 1, 1);
	int smemSize = blockSize.x * bpp;

	if (bpp == 3) {
		convert2greyscale_cuda_kernel_RGB<<<gridSize, blockSize>>>(width, height_parm, bpp, row_stride, frame_data_device, frame_data_device, ref_data_device);
		reduce6_RGB<BLOCK_SIZE_1D> <<<gridSize, blockSize, smemSize>>>(frame_data_device, devReduction, true_size / bpp);
		reduce6_int<BLOCK_SIZE_1D> <<<gridSize2, blockSize, smemSize>>>(devReduction, devReduction, gridSize.x);
	} else if (bpp == 4) {
		convert2greyscale_cuda_kernel_RGBx<<<gridSize, blockSize>>>(width, height_parm, bpp, row_stride, frame_data_device, frame_data_device, ref_data_device);
		reduce6_RGBx<BLOCK_SIZE_1D> <<<gridSize, blockSize, smemSize>>>(frame_data_device, devReduction, true_size / bpp);
		reduce6_int<BLOCK_SIZE_1D> <<<gridSize2, blockSize, smemSize>>>(devReduction, devReduction, gridSize.x);
	}

	hipMemcpy(data, frame_data_device, true_size, hipMemcpyDeviceToHost);
	hipMemcpy(reduction, devReduction, gridSize2.x * sizeof(int), hipMemcpyDeviceToHost);

	pixel_diff_gpgpu = 0;
	for (unsigned int i = 0; i < gridSize2.x; i++)
		pixel_diff_gpgpu += reduction[i];

	hipHostUnregister(data);

	return pixel_diff_gpgpu;
}
